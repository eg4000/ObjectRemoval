#include "hip/hip_runtime.h"
#include <cstdlib>
//#include <cstdio>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
// texture<float, 2, hipReadModeElementType> inTex;
texture<float4, 2, hipReadModeElementType> inTex;
// texture<float, hipTextureType1D, hipReadModeElementType> inTex; 

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

const int N_THREADS = 64;
const int N_BLOCKS = 64;

__global__ void copy_mem(unsigned char *source, unsigned char *render)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      for (int channel = 0; channel < 3; channel ++ )
        render[3*((y+j)*width + x) + channel] = source[3 * ((y+j)*width + x) + channel];
}


__global__ void set_depth(unsigned int *depth)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      depth[(y+j)*width + x] = 65535;
}


__global__ void char_to_int(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      img2[(y+j)*width + x] =  img[3*((y+j)*width + x) + 0] * 256 * 256 + img[3*((y+j)*width + x) + 1] * 256 + img[3*((y+j)*width + x) + 2];
}


__global__ void int_to_char(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
      img[3*((y+j)*width + x)] = img2[(y+j)*width + x] / (256*256);
      img[3*((y+j)*width + x)+1] = img2[(y+j)*width + x] / 256 % 256;
      img[3*((y+j)*width + x)+2] = img2[(y+j)*width + x] % 256;
    }
}


__global__ void to3d_point(float *depth, float *points3d)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w / 2;
    
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     
     int iw = x;
     int ih = y + j;
     float depth_point = depth[ih*w + iw] * 128.0;
     float phi = ((float)(ih) + 0.5) / float(h) * M_PI;
     float theta = ((float)(iw) + 0.5) / float(w) * 2 * M_PI + M_PI;
  
      points3d[(ih * w + iw) * 4 + 0] = depth_point * sin(phi) * cos(theta);
      points3d[(ih * w + iw) * 4 + 1] = depth_point * sin(phi) * sin(theta);
      points3d[(ih * w + iw) * 4 + 2] = depth_point * cos(phi);
      points3d[(ih * w + iw) * 4 + 3] = 1;
  
  }
}

__global__ void transform(float *points3d_after, float *points3d, float * transformation_matrix)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     for (int ic = 0; ic < 3; ic ++) {
     points3d_after[(ih * w + iw) * 3 + ic] = points3d[(ih * w + iw) * 4 + 0] * transformation_matrix[4 * ic + 0]
     + points3d[(ih * w + iw) * 4 + 1] * transformation_matrix[4 * ic + 1] 
     + points3d[(ih * w + iw) * 4 + 2] * transformation_matrix[4 * ic + 2] 
     + points3d[(ih * w + iw) * 4 + 3] * transformation_matrix[4 * ic + 3]; 
    }
  }
}


__global__ void transform2d(float *points3d_after, float *points3d_polar)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     float x = points3d_after[(ih * w + iw) * 3 + 0];
     float y = points3d_after[(ih * w + iw) * 3 + 1];
     float z = points3d_after[(ih * w + iw) * 3 + 2];

    points3d_polar[(ih * w + iw) * 3 + 0] = sqrt(x * x + y * y + z * z);
    points3d_polar[(ih * w + iw) * 3 + 1] = atan2(y, x);
    points3d_polar[(ih * w + iw) * 3 + 2] = atan2(sqrt(x * x + y * y), z);
  }
}


__global__ void render_depth(float *points3d_polar, unsigned int * depth_render)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     int this_depth = (int)(512 * points3d_polar[(ih * w + iw) * 3 + 0]);
     atomicMin(&depth_render[(ty * w + tx)] , this_depth);
  }
}



__global__ void render_final(float *points3d_polar, int * depth_render, int * img,  int * render)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     int this_depth = (int)(12800/128 * points3d_polar[(ih * w + iw) * 3 + 0]);
     int delta = this_depth - depth_render[(ty * w + tx)];
     
     //printf("%d %d\n", this_depth, depth_render[(ty * w + tx)]);
     if ((y > h/8) && (y < h*7/8))
     if ((delta > -10) && (delta < 10) && (this_depth < 10000)) {
           render[(ty * w + tx)] = img[(ih * w + iw)];
     }
  }
}


__global__ void projectCubeMapToERImage(float *dst, float * src, uint * idxs,  size_t count)
{
  int n_to_do = count / ( gridDim.x * blockDim.x);
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * n_to_do;
  //printf("x: %d w: %d | %d %d (%d)(%d)\n", blockIdx.x, threadIdx.x, gridDim.x, blockDim.x, start, n_to_do);
  for (int j = 0; j < n_to_do; j++)
  {
    dst[start + j] = src[idxs[start + j]];
  }
}

__global__ void readTextureToCubeMapBuffer(float * dst, size_t width, size_t height)
{
    unsigned int n_to_do = height * width / (blockDim.x * gridDim.x);
    int start = (blockIdx.x * blockDim.x + threadIdx.x) * n_to_do;
    // printf("Block (%i) thread (%i); n_to_do (%d); start (%d) | (%d, %d)\n", 
    //         blockIdx.x, threadIdx.x , 
    //         n_to_do, 
    //         start, width, height);
    for (int j = start; j < start + n_to_do; j++)
    {
        int x_val = (j%width);
        int y_val = (j/width);
        float4 temp = tex2D(inTex, x_val, y_val); ;
        dst[j] = temp.z;
    }
    // printf("DONE (%i, %i);\n", blockIdx.x, threadIdx.x);
}



extern "C"{

/* Convenience function to print any GPU errors */
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

float * allocateBufferOnGPU(size_t count)
{
    float *d_dst;
    const int dst_mem_size = count*sizeof(float);
    hipMalloc((void **)&d_dst, dst_mem_size);
    return d_dst;
}

void projectCubeMapToEquirectangular(float * dst, float * d_src, uint *d_idx, size_t count, size_t src_size)
{
    /* First call "d_idx = moveToGPU(cubeIdxToEqui, count)" */

    // Declare vars
    const int dstMemSize = count*sizeof(float);
    float *d_dst;

    // Create buffer for the equirectangular img on gpu
    hipMalloc((void **)&d_dst, dstMemSize);    
    hipMemcpy(d_dst, dst, dstMemSize, hipMemcpyHostToDevice);

    // Do cube -> equirecangular projection
    projectCubeMapToERImage<<< N_BLOCKS, N_THREADS >>>(d_dst, d_src, d_idx, count);
    
    // Copy back to cpu
    hipMemcpy(dst, d_dst, dstMemSize, hipMemcpyDeviceToHost);

    hipFree(d_dst);
    hipDeviceSynchronize();
}

void fillBlue(float * dst, hipArray_t src, size_t offset, size_t w, size_t h)
{
    /* Fills the buffer at *dst with the contents at src + offset (a h x w texture)*/
    // --- Dims
    dim3 dimBlock(N_BLOCKS);
    dim3 dimGrid(N_THREADS);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    // Set the texture parameters
    inTex.normalized = false;
    hipBindTextureToArray(inTex, src, channelDesc); 
    readTextureToCubeMapBuffer<<< dimBlock, dimGrid >>>(dst + offset, w, h);
}

uint * copyToGPU(uint * cubeMapIdxToEqui, size_t count) 
{
    /* Copies the given array to device */
    uint *d_idx;
    const int idxsMemSize = count*sizeof(uint);
    hipMalloc((void **)&d_idx, idxsMemSize);
    hipMemcpy(d_idx, cubeMapIdxToEqui, idxsMemSize, hipMemcpyHostToDevice);
    return d_idx;
}

void render(int h,int w,unsigned char * img, float * depth,float * pose, unsigned char * render, int * depth_render)
{
    //int ih, iw, i, ic;
    
    const int nx = w;
    const int ny = h;
    const int depth_mem_size = nx*ny*sizeof(float);
    const int frame_mem_size = nx*ny*sizeof(unsigned char) * 3;
    
    dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
    
    unsigned char *d_img, *d_render;
    float *d_depth, *d_pose;
    int *d_depth_render;
    float *d_3dpoint, *d_3dpoint_after, *d_3dpoint_polar;
    
    int *d_render2, *d_img2;
    
    hipMalloc((void **)&d_img, frame_mem_size);
    hipMalloc((void **)&d_render, frame_mem_size);
    hipMalloc((void **)&d_depth, depth_mem_size);
    hipMalloc((void **)&d_depth_render, nx * ny * sizeof(int));
    hipMalloc((void **)&d_3dpoint, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_after, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_polar, depth_mem_size * 4);
    hipMalloc((void **)&d_pose, sizeof(float) * 16);
    hipMalloc((void **)&d_render2, nx * ny * sizeof(int));
    hipMalloc((void **)&d_img2, nx * ny * sizeof(int));
    
    hipMemcpy(d_depth_render, depth_render, nx * ny * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pose, pose, sizeof(float) * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_img, img, frame_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_depth, depth, depth_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_render, render, frame_mem_size, hipMemcpyHostToDevice);
    
    hipMemset(d_render2, 0, nx * ny * sizeof(int));
    hipMemset(d_img2, 0, nx * ny * sizeof(int));
    
    hipMemset(d_3dpoint, 0, depth_mem_size * 4);
    hipMemset(d_3dpoint_after, 0, depth_mem_size * 4);
    
    to3d_point<<< dimGrid, dimBlock >>>(d_depth, d_3dpoint);
    transform<<< dimGrid, dimBlock >>>(d_3dpoint_after, d_3dpoint, d_pose);
    transform2d<<<dimGrid, dimBlock>>>(d_3dpoint_after, d_3dpoint_polar);
    
    char_to_int <<< dimGrid, dimBlock >>> (d_img2, d_img);
    char_to_int <<< dimGrid, dimBlock >>> (d_render2, d_render);
    
    //render_depth <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render);
    render_final <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render, d_img2, d_render2);
    
    int_to_char <<< dimGrid, dimBlock >>> (d_render2, d_render);
    
    hipMemcpy(render, d_render, frame_mem_size, hipMemcpyDeviceToHost);
    //hipMemcpy(depth_render, d_depth_render, nx * ny * sizeof(unsigned int), hipMemcpyDeviceToHost);
        
    hipFree(d_img);
    hipFree(d_depth);
    hipFree(d_render2);
    hipFree(d_img2);
    hipFree(d_render);
    hipFree(d_depth_render);
    hipFree(d_3dpoint);
    hipFree(d_3dpoint_after);
    hipFree(d_3dpoint_polar);
    hipFree(d_pose);
}

}//extern "C"